#include "hip/hip_runtime.h"
#include "kernal.cuh"
#include <cmath>

/*__device__ template <typename T> int sgn(T val) {
	return (T(0) < val) - (val < T(0));
}*/

//TODO CHANGE EVERYTHING TO POINTERS, WHY DIDNT I DO THAT IN THE FIRST PLACE
//TODO GET RID OFF ALL THE VECTOR3 TO FLOAT3 TRANSLATIONS

__device__ bool CheckBounding(float3 nPos, float aggroRange, float3 pos, float3 halfDim)
{
	float dist = abs(pos.x - nPos.x);
	float sum = halfDim.x + aggroRange;

	if(dist <= sum) {
		dist = abs(pos.y - nPos.y);
		sum = halfDim.y + aggroRange;

		if(dist <= sum) {
			dist = abs(pos.z - nPos.z);
			sum = halfDim.z + aggroRange;

			if(dist <= sum) {
				//if there is collision data storage
				return true;
			}
		}
	}
	return false;
}

__device__ void cudaPatrol(Players* players, Agents* agents, float msec, const unsigned int size)
{
	int a = blockIdx.x * blockDim.x + threadIdx.x;

	float MAXSPEED = 0.5F;

	//at target
	float diffX = agents->patrolLocation[a][agents->targetLocation[a]].x - agents->x[a];
	float diffZ = agents->patrolLocation[a][agents->targetLocation[a]].z - agents->z[a];
	float absX = abs(diffX);
	float absZ = abs(diffZ);

	//check its close enough to the point
	if (absX < 0.1f && absZ < 0.1f)
	{
		//get new target
		agents->targetLocation[a]++;
		agents->targetLocation[a] = agents->targetLocation[a] % 2; //need to fix this
	}
	else
	{
		//move to target
		float dis = absX + absZ;
		float moveX = ((absX / dis) * MAXSPEED) * msec;
		float moveZ = ((absZ / dis) * MAXSPEED) * msec;

		//find how much it needs to move
		moveX = min(moveX, absX);
		moveZ = min(moveZ, absZ);

		//set new position
		agents->x[a] += moveX * ((float(0) < diffX) - (diffX < float(0)));
		agents->z[a] += moveZ * ((float(0) < diffZ) - (diffZ < float(0)));
	}

	//state transition

	int i = 0;
	// loop through all the players
	/*while (i < players->MAXPLAYERS && agents->players[a][i] > -1)
	{

		//the player
		int p = agents->players[a][i];
		//calculate distance to player
		float3 diff = float3();

		diff.x = players->x[p] - agents->x[a];
		diff.y = players->y[p] - agents->y[a];
		diff.z = players->z[p] - agents->z[a];

		float dist = sqrtf((diff.x*diff.x)+(diff.y*diff.y)+(diff.z*diff.z));
	


		//if player close transition state to stare at player
		float aggroRange = min(agents->AGGRORANGE, agents->AGGRORANGE * ((float)agents->level[a] / (float)players->level[p]));

		if (dist < aggroRange && !players->isDead[p])
		{
			agents->state[a] = STARE_AT_PLAYER; //change state
			agents->patrolLocation[a][2].x = agents->x[a];
			agents->patrolLocation[a][2].y = agents->y[a];
			agents->patrolLocation[a][2].z = agents->z[a]; //set position it left patrol
			agents->targetPlayer[a] = p; // playing that is being stared at
			i = players->MAXPLAYERS; // exit the loop
		}
		i++;
	}*/
}

__device__ void cudaStareAtPlayer(Players* players, Agents* agents, float msec, const unsigned int size)
{
	int a = blockIdx.x * blockDim.x + threadIdx.x;

	int p = agents->targetPlayer[a]; // target player

	//calculate distance to player
	float3 diff = float3();
	diff.x = players->x[p] - agents->x[a];
	diff.y = players->y[p] - agents->y[a];
	diff.z = players->z[p] - agents->z[a];
	float dist = sqrtf((diff.x*diff.x)+(diff.y*diff.y)+(diff.z*diff.z));

	//the range of aggro, and pull, to the player
	float aggroRange = min(agents->AGGRORANGE, agents->AGGRORANGE * ((float)agents->level[a] / (float)players->level[p]));
	float pullRange = (aggroRange * 0.75f) * ((float)agents->level[a] / (float)players->level[p]);


	if (dist < pullRange && !players->isDead[p]) // if the player is in pull range
	{
		agents->state[a] = CHASE_PLAYER;
	}
	else
	{
		// if the player isnt in pull range check if there are any players closer
		bool playerClose = false;
		int i = 0;

		//loop through the players
		while (i < players->MAXPLAYERS && agents->players[a][i] > -1)
		{
			int p2 = agents->players[a][i];

			//calculate distance to player
			float3 diffNew = float3();
			diffNew.x = players->x[p2] - agents->x[a];
			diffNew.y = players->y[p2] - agents->y[a];
			diffNew.z = players->z[p2] - agents->z[a];
			float distNew = sqrtf((diffNew.x*diffNew.x)+(diffNew.y*diffNew.y)+(diffNew.z*diffNew.z));

			// if the new distance is less switch targte
			if (distNew <= dist  && !players->isDead[p2])
			{
				agents->targetPlayer[a] = p2;
				dist = distNew;
				float aggroRangeNew = min(agents->AGGRORANGE, agents->AGGRORANGE * (agents->level[a] / players->level[p2]));

				if (dist < aggroRangeNew)
				{
					playerClose = true;
				}
			}
			++i;
		}

		// if there are no close players at all
		if (!playerClose)
		{
			agents->state[a] = PATROL;
			agents->targetPlayer[a] = -1;
		}
	}
}

__device__ void cudaChasePlayer(Players* players, Agents* agents, float msec, const unsigned int size)
{
	int a = blockIdx.x * blockDim.x + threadIdx.x;
	
	float LEASHRANGE = 3200.0f;
	float ATTACKRANGE = 75.0f;
	float MAXSPEED = 0.5F;

	int p = agents->targetPlayer[a];

	//calculate distance to leash spot
	float3 diff = float3();
	diff.x = agents->patrolLocation[a][2].x - agents->x[a];
	diff.y = agents->patrolLocation[a][2].y - agents->y[a];
	diff.z = agents->patrolLocation[a][2].z - agents->z[a];

	float leashDist = sqrtf((diff.x*diff.x)+(diff.y*diff.y)+(diff.z*diff.z));;

	// if its too far away or if the player died leash back
	if (leashDist > LEASHRANGE || players->isDead[p])
	{
		agents->state[a] = LEASH;
		agents->targetPlayer[a] = -1;
	}
	else
	{
		//calculate distance to player
		float3 diff = float3();
		diff.x = players->x[p] - agents->x[a];
		diff.y = players->y[p] - agents->y[a];
		diff.z = players->z[p] - agents->z[a];
		float dist = sqrtf((diff.x*diff.x)+(diff.y*diff.y)+(diff.z*diff.z));

		//if close to player switch state to useability
		if (dist < ATTACKRANGE)
		{
			agents->state[a] = USE_ABILITY;
		}

		//move towards players location
		float absX = abs(diff.x);
		float absZ = abs(diff.z);

		//move to target
		float dis = absX + absZ;
		float moveX = ((absX / dis) * MAXSPEED) * msec;
		float moveZ = ((absZ / dis) * MAXSPEED) * msec;

		moveX = min(moveX, absX);
		moveZ = min(moveZ, absZ);

		//set new position
		agents->x[a] += moveX * ((float(0) < diff.x) - (diff.x < float(0)));
		agents->z[a] += moveZ * ((float(0) < diff.z) - (diff.z < float(0)));
	}

}

__device__ void cudaLeashBack(Players* players, Agents* agents, float msec, const unsigned int size)
{
	int a = blockIdx.x * blockDim.x + threadIdx.x;

	float MAXSPEED = 0.5F;

	//calculate distance to leash spot
	float diffX = agents->patrolLocation[a][2].x - agents->x[a];
	float diffZ = agents->patrolLocation[a][2].z - agents->z[a];
	float absX = abs(diffX);
	float absZ = abs(diffZ);

	//check its close enough to the point
	if (absX < 0.1f && absZ < 0.1f)
	{
		//change back to patrol
		agents->state[a] = PATROL;
	}
	else
	{
		//move to target
		float dis = absX + absZ;
		float moveX = ((absX / dis) * MAXSPEED) * msec;
		float moveZ = ((absZ / dis) * MAXSPEED) * msec;

		moveX = min(moveX, absX);
		moveZ = min(moveZ, absZ);

		//set new position
		agents->x[a] += moveX * ((float(0) < diffX) - (diffX < float(0)));
		agents->z[a] += moveZ * ((float(0) < diffZ) - (diffZ < float(0)));
	}
}

__device__ void cudaUseAbility(Players* players, Agents* agents, float msec, const unsigned int size)
{
	int a = blockIdx.x * blockDim.x + threadIdx.x;

	float ATTACKRANGE = 75.0f;
	int p = agents->targetPlayer[a];

	if (players->isDead[p]) // if the player is dead
	{
		agents->state[a] = LEASH;	//leash back
		agents->targetPlayer[a] = -1; // set the target player to null
	}
	else
	{

		//TODO ADD ABILITIES BACK
		//look through abilities via priority until one is found not on cooldown
		int i = 0;
		while (i < agents->MAXABILITIES && agents->myAbilities[a][i].cooldown > 0.001f) {
			i++;
		}

		//cast ability
		if (i < agents->MAXABILITIES && agents->myAbilities[a][i].cooldown < 0.001f)
		{
			agents->myAbilities[a][i].cooldown = agents->myAbilities[a][i].maxCooldown;
			players->hp[agents->targetPlayer[a]] -= agents->myAbilities[a][i].damage;
		}

		//if the player goes out of range, change state to chase
		//calculate distance to player
		float3 diff = float3();
		diff.x = players->x[p] - agents->x[a];
		diff.y = players->y[p] - agents->y[a];
		diff.z = players->z[p] - agents->z[a];
		float dist = sqrtf((diff.x*diff.x)+(diff.y*diff.y)+(diff.z*diff.z));

		//if player close transition state to stare at player
		if (dist > (ATTACKRANGE))
		{
			agents->state[a] = CHASE_PLAYER;
		}
	}
}

__global__ void cudaBroadphasePlayers(Players* players, AIWorldPartition* partitions, const int partitionCount, Vector3* halfDim)
{
	int pa = blockIdx.x * blockDim.x + threadIdx.x;

	//Vector3 to float3 convertions
	/*float3 hDim = float3();
	hDim.x = halfDim->x;
	hDim.y = halfDim->y;
	hDim.z = halfDim->z;

	float3 pPos = float3();
	pPos.x = partitions[pa].pos.x;
	pPos.y = partitions[pa].pos.y;
	pPos.z = partitions[pa].pos.z;

	for (int i = 0; i < players->MAXPLAYERS; ++i)
	{
		//get player pos in float3
		float3 nPos = float3();
		nPos.x = players->x[i];
		nPos.y = players->y[i];
		nPos.z = players->z[i];

		if (CheckBounding(nPos, 0, pPos, hDim))
		{
			partitions[pa].myPlayers[partitions[pa].playerCount] = i;
			++partitions[pa].playerCount;
		}
	}*/

}

__global__ void cudaBroadphaseAgents(Agents* agents, AIWorldPartition* partitions, const int partitionCount, Vector3* halfDim)
{
	int a = blockIdx.x * blockDim.x + threadIdx.x;

	//Vector3 to float3 convertions
	/*float3 hDim = float3();
	hDim.x = halfDim->x;
	hDim.y = halfDim->y;
	hDim.z = halfDim->z;

	float3 nPos = float3();
	nPos.x = agents->x[a];
	nPos.y = agents->y[a];
	nPos.z = agents->z[a];

	//loop through the partitions
	for (int i = 0; i < partitionCount; ++i)
	{
		float3 pPos = float3();
		pPos.x = partitions[i].pos.x;
		pPos.y = partitions[i].pos.y;
		pPos.z = partitions[i].pos.z;

		//if the agent is in the partition
		if (CheckBounding(nPos, agents->AGGRORANGE, pPos, hDim)) {
			//loop through all the players and copy them to the agent
			for (int j = 0; j < partitions[i].playerCount; ++j)
			{
				agents->players[a][j] = partitions[i].myPlayers[j];
			}
		}
	}*/
}

__global__ void cudaFSM(Players* players, Agents* agents, float msec, const unsigned int size)
{
	int a = blockIdx.x * blockDim.x + threadIdx.x;

	switch (agents->state[a]) {
	case PATROL: cudaPatrol(players, agents, msec, size);
		break;
	case STARE_AT_PLAYER: cudaStareAtPlayer(players, agents, msec, size);
		break;
	case CHASE_PLAYER: cudaChasePlayer(players, agents, msec, size);
		break;
	case LEASH: cudaLeashBack(players, agents, msec, size);
		break;
	case USE_ABILITY: cudaUseAbility(players, agents, msec, size);
		break;
	};

}


//put the data onto the GPU
hipError_t addDataToGPU(Players* players, Agents* agents, unsigned int size, float msec, Players* d_players, Agents* d_agents)
{
    d_players = 0;
	d_agents = 0;
	//Agents* dev_agents = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		return cudaStatus;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&d_agents, sizeof(Agents));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed 1!");
		return cudaStatus;
    }

    cudaStatus = hipMalloc((void**)&d_players, sizeof(Players));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed! 2");
		return cudaStatus;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(d_agents, agents, sizeof(Agents), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed! 3");
		return cudaStatus;
    }

    cudaStatus = hipMemcpy(d_players, players, sizeof(Players), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed! 4");
		return cudaStatus;
    }

	return cudaStatus;
}

hipError_t runKernal(Players* players, Agents* agents, unsigned int size, float msec, Players* d_players, Agents* d_agents)
{
	hipError_t cudaStatus;
	int blockSize;      // The launch configurator returned block size 
    int minGridSize;    // The minimum grid size needed to achieve the maximum occupancy for a full device launch 
    int gridSize;       // The actual grid size needed, based on input size

	hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, cudaFSM, 0, size);

	// Round up according to array size 
	gridSize = (size + blockSize - 1) / blockSize;

	//hipOccupancyMaxActiveBlocksPerMultiprocessor(&minGridSize, cudaPatrol, blockSize, 0);

	// Launch a kernel on the GPU with one thread for each element.
	cudaFSM<<<gridSize, blockSize>>>(d_players, d_agents, msec, size);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		return cudaStatus;
    }

	// hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		fprintf(stderr, hipGetErrorName(cudaStatus));
    }

	return cudaStatus;
}

hipError_t clearData(Players* players, Agents* agents, unsigned int size, float msec, Players* d_players, Agents* d_agents)
{
	hipError_t cudaStatus;

    // Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(players, d_players, sizeof(Players), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed players!");
        goto Error;
    }

	cudaStatus = hipMemcpy(agents, d_agents, sizeof(Agents), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed agents!");
        goto Error;
    }


Error:
    //hipFree(dev_c);
    hipFree(d_agents);
    hipFree(d_players);
   

    return cudaStatus;
}

hipError_t addWithCuda(Players* players, Agents* agents, const unsigned int size, float msec, Players* d_players, Agents* d_agents)
{
    d_players = 0;
	d_agents = 0;
    hipError_t cudaStatus;
	int blockSize;      // The launch configurator returned block size 
    int minGridSize;    // The minimum grid size needed to achieve the maximum occupancy for a full device launch 
    int gridSize;       // The actual grid size needed, based on input size 


    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&d_agents, sizeof(Agents));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&d_players, sizeof(Players));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(d_agents, agents, sizeof(Agents), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(d_players, players, sizeof(Players), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

	hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, cudaFSM, 0, size);

	// Round up according to array size 
	gridSize = (size + blockSize - 1) / blockSize;

	//hipOccupancyMaxActiveBlocksPerMultiprocessor(&minGridSize, cudaPatrol, blockSize, 0);

	// Launch a kernel on the GPU with one thread for each element.
	cudaFSM<<<gridSize, blockSize>>>(d_players, d_agents, msec, size);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(players, d_players, sizeof(Players), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed players!");
        goto Error;
    }

	cudaStatus = hipMemcpy(agents, d_agents, sizeof(Agents), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed agents!");
        goto Error;
    }


Error:
    //hipFree(dev_c);
    hipFree(d_agents);
    hipFree(d_players);
   

    return cudaStatus;
}

hipError_t cudaUpdateAgents(Players* players, Agents* agents, const unsigned int size, float msec, AIWorldPartition* partitions, const int partitionCount, Vector3* halfDim)
{
	//COPY DATA TO THE GPU
	//////////////////////

    Players* d_players = 0;
	Agents* d_agents = 0;
	AIWorldPartition* d_partitions = 0;
	Vector3* d_halfDim = 0;
    hipError_t cudaStatus;
	int blockSize;      // The launch configurator returned block size 
    int minGridSize;    // The minimum grid size needed to achieve the maximum occupancy for a full device launch 
    int gridSize;       // The actual grid size needed, based on input size 


    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for the data
	// Agents
    cudaStatus = hipMalloc((void**)&d_agents, sizeof(Agents));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

	// Players
    cudaStatus = hipMalloc((void**)&d_players, sizeof(Players));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

	// World Partitions
	cudaStatus = hipMalloc((void**)&d_partitions, partitionCount * sizeof(AIWorldPartition));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

	// Half Dimensions
	cudaStatus = hipMalloc((void**)&d_halfDim, sizeof(Vector3));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(d_agents, agents, sizeof(Agents), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(d_players, players, sizeof(Players), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

	cudaStatus = hipMemcpy(d_partitions, partitions, sizeof(AIWorldPartition), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

	cudaStatus = hipMemcpy(d_halfDim, halfDim, sizeof(Vector3), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

	//RUN THE KERNALS ON THE GPU
	////////////////////////////

	//get the mingrid and blocksize
	/*hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, cudaBroadphasePlayers, 0, partitionCount);

	// Round up according to array size 
	gridSize = (size + blockSize - 1) / blockSize;

	// Launch a kernel on the GPU with one thread for each element.
	cudaBroadphasePlayers<<<gridSize, blockSize>>>(d_players, d_partitions, partitionCount, halfDim);

	cudaStatus = hipDeviceSynchronize();
	cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

	///////////////////////////

	//get the mingrid and blocksize
	hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, cudaBroadphaseAgents, 0, size);

	// Round up according to array size 
	gridSize = (size + blockSize - 1) / blockSize;

	// Launch a kernel on the GPU with one thread for each element.
	cudaBroadphaseAgents<<<gridSize, blockSize>>>(d_agents, d_partitions, partitionCount, halfDim);

	cudaStatus = hipDeviceSynchronize();
	cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }*/

	//////////////////////////

	//get the mingrid and blocksize
	hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, cudaFSM, 0, size);

	// Round up according to array size 
	gridSize = (size + blockSize - 1) / blockSize;

	// Launch a kernel on the GPU with one thread for each element.
	cudaFSM<<<gridSize, blockSize>>>(d_players, d_agents, msec, size);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }


	//COPY THE DATA BACK OFF OF THE GPU
	///////////////////////////////////

    // Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(players, d_players, sizeof(Players), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed players!");
        goto Error;
    }

	cudaStatus = hipMemcpy(agents, d_agents, sizeof(Agents), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed agents!");
        goto Error;
    }


Error:
    //hipFree(dev_c);
    hipFree(d_agents);
    hipFree(d_players);
   

    return cudaStatus;
}